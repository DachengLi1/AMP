#include "hip/hip_runtime.h"


#include "custom_cuda_layers.h"

__global__ void param_update_kernel(const float* input, __half* output, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < size) { output[id] = (__half)input[id]; }
}

void launch_param_update(const float* input, __half* output, int size, hipStream_t stream)
{
    int threads = 1024;

    dim3 grid_dim((size - 1) / threads + 1);
    dim3 block_dim(threads);

    param_update_kernel<<<grid_dim, block_dim, 0, stream>>>(input, output, size);
}
